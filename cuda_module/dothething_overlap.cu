#include "hip/hip_runtime.h"
#include "cuda_process.cuh"
#include "cuda_module.h"
#include "hip/hip_runtime.h"
#include ""
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"

#include <stdio.h>
#include <iostream>
#include <hipfft/hipfft.h>
#include <complex>

//int main(int argc, char **argv){}

//The cufft must be invoked by the host, not as part of a kernel. 
//num_wins is the total number of windows, without averaging. Therefore total windows abould be number of output frames * averaging to ensure no wasted samples.
float* dothething_overlap(std::complex<short>* h_samp_arry, const int averaging, float* h_out, const int num_wins, const int overlap) {

	bool error = false;
	/*
	if (num_wins == 0) {
		std::cout << "AMG NO WINS!\n";
		num_wins = sizeof(h_samp_arry) / (2 * NUM_SAMPS);
		std::cout << "Number of windows: " << num_wins << std::endl;
	} */
	//const int num_wins = 1;
	//hipComplex* samp[NUM_SAMPS];
	//std::complex<short>* d_samp;

	hipError_t cudaStatus;

	// for outputting of averaged and processed samples /
	
	h_out = (float*)malloc((sizeof(float) * NUM_SAMPS * (num_wins*overlap)/averaging));
	//h_out = (float*)calloc(NUM_SAMPS * num_wins / averaging, sizeof(float));
	if (h_out == NULL) {
		fprintf(stderr, "h_out Malloc failed!");
		goto Error;
	}
	
	float* d_out;
	
	cuComplexShort* h_samp_ptr = (cuComplexShort*)&h_samp_arry[0];

	//std::cout << h_samp_arry[0].real() << "," << h_samp_arry[0].imag() << " cuCmplx" << h_samp_ptr[0].x << "," << h_samp_ptr[0].y << std::endl;
	
	float h_win[NUM_SAMPS];
	float* d_win;

	cuComplexShort* d_samp;
	hipComplex* d_fftbuff;

	float win_power = 0;
	int rx_gain = 30;

	//Create coefficient array and x axis index for plotting
	for (int i = 0; i < NUM_SAMPS; i++) {
		h_win[i] = 0.35875 - 0.48829*cos(2 * pi*i / (NUM_SAMPS - 1)) + 0.14128*cos(4 * pi*i / (NUM_SAMPS - 1)) - 0.01168*cos(6 * pi*i / (NUM_SAMPS - 1)); //blackmann harris window		
		win_power += (h_win[i] * h_win[i]); //this computes the total window power and normalises it to account for DC gain due to the window.
	}
	win_power /= NUM_SAMPS; //normalise the total window power across each sample.

	const float offset = -10 - rx_gain + 10 * std::log10(win_power); //-10 is the MAX power detected by the ADC and take into account the gain of the frontend.

	//printf("GPU Offset: %f", offset);

	//allocate the memory for the GPU
	cudaStatus = hipMalloc((float**)&d_out, (sizeof(float)*NUM_SAMPS * (num_wins*overlap) / averaging));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc d_out failed!");
		error = true;
		goto Error;
	}
	hipMemset(d_out, 0, (sizeof(float)*NUM_SAMPS * (num_wins*overlap) / averaging)); //initialise to zero

	cudaStatus = hipMalloc((cuComplexShort**)&d_samp, sizeof(cuComplexShort)*NUM_SAMPS*num_wins);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc d_samp failed!");
		error = true;
		goto Error;
	}
	
	cudaStatus = hipMalloc((float**)&d_win, sizeof(float)*NUM_SAMPS);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc d_win failed!");
		error = true;
		goto Error;
	}
	
	cudaStatus = hipMalloc((hipComplex**)&d_fftbuff, sizeof(hipComplex)*NUM_SAMPS*num_wins*overlap);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc d_fftbuff failed!");
		error = true;
		goto Error;
	}
	
	//Transfer data to GPU
	cudaStatus = hipMemcpy(d_win, h_win, sizeof(float)*NUM_SAMPS, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy win to Device failed!");
		error = true;
		goto Error;
	}
	
	cudaStatus = hipMemcpy(d_samp, h_samp_ptr, sizeof(cuComplexShort)*NUM_SAMPS*num_wins, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy samp to Device failed!");
		error = true;
		goto Error;
	}
	
	//Create cufft plan, turns out cufft handles its own memory transfers, so we must use callbacks in order to avoid numerous reads and writes in the device
	//Will however use multiple kernels initially, then see what the performance improvement is with callbacks at a later stage. n.n
	hipfftHandle plan;
	hipfftPlan1d(&plan, NUM_SAMPS, HIPFFT_C2C, num_wins*overlap);

	//printf("h_samp[%d]=%f,%f ", 0, s_ptr[0].x, s_ptr[0].y);
	//printf("d_samp[%d]=%f,%f\n", 0,d_samp[0].x,d_samp[0].y);
	// Kernel calls lah <<blocks,threads>>
	
	cufft_prep_overlap<<<(NUM_SAMPS*num_wins) / CU_THD, CU_THD >>>(d_fftbuff, d_samp, d_win, num_wins, overlap); //This will create (WIN_SAMPS*num_wins)/CU_THD blocks, with 1024 threads per block
	
	//inplace fft
	if (hipfftExecC2C(plan, d_fftbuff, d_fftbuff, HIPFFT_FORWARD)){
		fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
		error = true;
		goto Error;
	}
	
	//Do something with the fft'd samples, like average them, then output them to the host, where the host can perform detection.
	avg_out_overlap <<<NUM_SAMPS / CU_THD, CU_THD >>>(d_out, d_fftbuff, num_wins, averaging, offset, overlap);
	
	cudaStatus = hipMemcpy(h_out, d_out, sizeof(float)*NUM_SAMPS * ((num_wins*overlap)/averaging), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy out to Host failed!");
		error = true;
		goto Error;
	}
	
	/*
	std::cout << "GPU: ";
	
	for (int i = 0; i < NUM_SAMPS; i++) {
		std::cout << h_win[i] << ",";
	}
	
	std::cout << "Please note these are not flipped around samples/2 correctly" << std::endl;
	*/

Error:

	hipfftDestroy(plan);
	checkCudaErrors(hipFree(d_out));
	checkCudaErrors(hipFree(d_samp));
	checkCudaErrors(hipFree(d_win));
	checkCudaErrors(hipFree(d_fftbuff));

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		error = true;
	}

	if (error) {
		fprintf(stderr, "\nGPU ERROPR!\n");
		system("pause");
	}

	return h_out;
}

//Kernel Call
//https://devblogs.nvidia.com/parallelforall/cuda-pro-tip-use-cufft-callbacks-custom-data-processing/ for inspiration
static __global__ void cufft_prep_overlap(hipComplex* d_fft, cuComplexShort* d_s, float* d_w, const int num_wins, int overlap) {

	int idx = threadIdx.x;
	
	//blockDim = number of threads in a block
	//This will take an array of complex shorts (14b samples) an array of hipComplex and a window array, will convert the com_short to hipComplex (com_float), correctly scale the samples and apply the appropriate window prepping it for fft
	for (int i = blockIdx.x * blockDim.x + idx, int j = blockIdx.x * blockDim.x + idx; i < NUM_SAMPS*num_wins*overlap; i += blockDim.x * gridDim.x, j += (blockDim.x * gridDim.x )/ overlap){
		d_fft[i].x = (d_s[j].x*1.0f / 32767.0f) * d_w[i%NUM_SAMPS];
		d_fft[i].y = (d_s[j].y*1.0f / 32767.0f) * d_w[i%NUM_SAMPS];
	}
	//if(idx == 0) printf("d_s[%d]: %f,%f fftbuff %f,%f\n", idx, d_s[idx].x, d_s[idx].y, d_s[idx].x, d_s[idx].x);
}

static __global__ void avg_out_overlap(float* out, hipComplex* d_fft, const int num_wins, const int averaging, const float offset, int overlap) {
	
	int idx = threadIdx.x;
	float* out_ptr = &out[0];
	hipComplex* d_fft_ptr = &d_fft[0];
	const float threshold = -113;

	for (int j = 0; j < (num_wins*overlap)/averaging; j++){

		for (int i = blockIdx.x * blockDim.x + idx; i < NUM_SAMPS*averaging; i += blockDim.x * gridDim.x){

			out_ptr[((NUM_SAMPS/2)+i)%NUM_SAMPS] += (
				10 * log10(abs(d_fft_ptr[i].x * d_fft_ptr[i].x + d_fft_ptr[i].y * d_fft_ptr[i].y) / NUM_SAMPS) //DFT bin magnitude
				);
		}

//		__syncthreads();

		out_ptr[(NUM_SAMPS / 2 + blockIdx.x * blockDim.x + idx) % NUM_SAMPS] = ((out_ptr[(NUM_SAMPS / 2 + blockIdx.x * blockDim.x + idx) % NUM_SAMPS] / averaging + offset) <= threshold) ? 1 : 0;
		//out_ptr[(NUM_SAMPS / 2 + blockIdx.x * blockDim.x + idx) % NUM_SAMPS] = (out_ptr[(NUM_SAMPS / 2 + blockIdx.x * blockDim.x + idx) % NUM_SAMPS] / averaging + offset);// <= threshold) ? 1 : 0;

//		if (out_ptr[blockIdx.x * blockDim.x + idx] <= threshold) out_ptr[blockIdx.x * blockDim.x + idx] = 1;
//		else out_ptr[blockIdx.x * blockDim.x + idx] = 0;

		out_ptr += NUM_SAMPS; //increment out_ptr by one frame of averages
		d_fft_ptr += NUM_SAMPS*averaging; //increment d_fft_ptr by number of frames averaged
	}
}

/* BACKUP LOL
static __global__ void avg_out(float* out, hipComplex* d_fft, const int num_wins, const int averaging) {

	int idx = threadIdx.x;
	float* out_ptr = &out[0];
	hipComplex* d_fft_ptr = &d_fft[0];

	for (int j = 0; j < num_wins / averaging; j++){

		for (int i = blockIdx.x * blockDim.x + idx; i < NUM_SAMPS*averaging; i += blockDim.x * gridDim.x){

			out_ptr[i%NUM_SAMPS] += (
				10 * log10(abs(d_fft_ptr[i].x * d_fft_ptr[i].x + d_fft_ptr[i].y * d_fft_ptr[i].y) / NUM_SAMPS) //DFT bin magnitude
				);
		}
		out_ptr += NUM_SAMPS; //increment out_ptr by one frame of averages
		d_fft_ptr += NUM_SAMPS*averaging; //increment d_fft_ptr by number of frames averaged
	}
}*/